#include <vector>
#include <iostream>
#include <set>
#include <algorithm>  
#include <fstream>
#include <stdlib.h>
#include <sstream>

#include "utils.h"
#include "helpers.h"



void parse_command_line(bool& show_assignment,
                        float& threshold,
                        std::string& matrix_file,
                        int argc,
                        char** argv,
                        bool& DEBUG) {
    int i = 1;
    while (i < argc) {
        std::string s(argv[i]);
        if (s == "-f") {
            matrix_file = std::string(argv[i + 1]);
            i += 2;
        } else if (s == "-g") {
            threshold = strtof(argv[i + 1], NULL);
            i += 2;
        } else if (s == "-v") {
            show_assignment = true;
            i += 1;
        }
        else if (s == "-d") {
            DEBUG = true;
            i += 1;
        } else {
            exit(1);
        }
    }
}

void print_clustering(int initial_n, vi& finalC) {
    std::vector<pi> finalCPrime;
    for (int i = 0; i < initial_n; ++i) {
        finalCPrime.push_back(pi(finalC[i], i));
    }
    
    std::cout << std::set<int>(finalC.begin(), finalC.end()).size();
    std::sort(finalCPrime.begin(), finalCPrime.end());
    
    int lastC = -1;
    for (auto& p : finalCPrime) {
        if (lastC != p.first) {
            std::cout << std::endl << p.first;
            lastC = p.first;
        }
        std::cout << " " << p.second;
    }
    std::cout << std::endl;
}

void read_graph_from_file( std::string& matrix_file, 
                        int& n,
                        int& m,
                        vi& V, 
                        vi& N,
                        vf& W) {    
    std::ifstream matrix_stream;
    matrix_stream.open(matrix_file);
    int entries = 0;
    
    std::string line;
    while (std::getline(matrix_stream, line)) {
        if (line[0] != '%') {
            std::stringstream(line) >> n >> n >> entries;
            break;
        }
    }    
    m = 0;
    std::vector<tr> tmp;
    for (int i = 0; i < entries; i++) {
        int v1, v2;
        float f;
        matrix_stream >> v1 >> v2 >> f;

        m++;
        tmp.push_back(tr(pi(v1 - 1,v2 - 1),f));
        if (v1 != v2) {
            m++;
            tmp.push_back(tr(pi(v2 - 1,v1 - 1),f));
        } 
    }
    
    //todo check if sorting is fast
    std::cerr << "starting sort" << std::endl;
    std::sort(tmp.begin(), tmp.end());
    std::cerr << "finished sort" << std::endl;

    V = vi(n + 1, 0);
    N = vi(m, 0);
    W = vf(m, 0);

    int v_idx = 0;
    for (size_t i = 0; i < tmp.size(); i++) {
        while (v_idx <= tmp[i].first.first) {
            V[v_idx++] = i;
        }
        N[i] = tmp[i].first.second;
        W[i] = tmp[i].second;
    }
    while (v_idx < n + 1) {
        V[v_idx++] = m;
    }
}

void start_recording_time(hipEvent_t& start_time, hipEvent_t& stop_time) {
    HANDLE_ERROR(hipEventCreate(&start_time));
    HANDLE_ERROR(hipEventCreate(&stop_time));
    HANDLE_ERROR(hipEventRecord(start_time, 0));
}

float stop_recording_time(hipEvent_t& start_time, hipEvent_t& stop_time) {
    float elapsed_time;
    HANDLE_ERROR(hipEventRecord(stop_time, 0));
    HANDLE_ERROR(hipEventSynchronize(stop_time));
    HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start_time, stop_time));
    HANDLE_ERROR(hipEventDestroy(start_time));
    HANDLE_ERROR(hipEventDestroy(stop_time));
    return elapsed_time;
}
